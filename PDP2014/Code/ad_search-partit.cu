#include "hip/hip_runtime.h"
/*
 *  Adaptive search - GPU Version
 *
 *
 * Please visit the https://pauillac.inria.fr/~diaz/adaptive/manual/index.html for a complete version of the original Adaptive Search code
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include<hiprand/hiprand_kernel.h>
#include <time.h>

#include "mrand.cu"


#define BIG ((unsigned int) -1 >> 1) 

#define BASE_MARK    ((unsigned) p_ad.nb_swap)
//#define Mark(i, k)   p_ad.mark[i] = BASE_MARK + (k)
#define Mark(i,k) {}
#define UnMark(i)    p_ad.mark[i] = 0
#define Marked(i)    (BASE_MARK + 1 <= p_ad.mark[i])

#define USE_PROB_SELECT_LOC_MIN ((unsigned) p_ad.prob_select_loc_min <= 100)

#define Div_Round_Up(x, y)   (((x) + (y) - 1) / (y))

#ifndef SIZE
#define SIZE 10
#endif

#ifndef THREADS
#define THREADS 2
#endif

//#define Arrayaccess(a, i, j) ((a)[(i) * SIZE + (j)])
#define Arrayaccess(i,j) ( (i) * SIZE + (j) )

typedef struct
{
  int i, j;
}Pair;

#ifndef GLOBALMEM //using global memory to store some variables
#define GLOBALMEM
#endif

typedef struct {
    int max_i;
    int min_j;
    int new_cost;
    int best_cost;
     //int *mark;
#ifndef GLOBALMEM
    int mark[SIZE];
#else
    int *mark;
#endif
    int nb_var_marked;
    int size;
    int seed;

    //int list_i[SIZE];
    int *list_i;
    int list_i_nb;

    //int list_j[SIZE];
    int *list_j;
    int list_j_nb;

    //int list_i2_nb[THREADS];
    //int list_j2_nb[THREADS];
    /*int *list_j2[THREADS];
    int *list_i2[THREADS];*/
    //int **list_j2;
    //int **list_i2;
#ifndef PAR
    int list_ij_nb;
    Pair list_ij[ SIZE ];
#else
    int list_ij2_nb[THREADS];
    int **list_ij2;
#endif
    int sol[SIZE];
    //int *sol;
    int total_cost;
    
    int nb_iter;
    int nb_iter_tot;
    int restart_limit;
    int nb_restart;
    int nb_swap;
    int nb_swap_tot;
    int swap;
    int nb_same_var;
    int nb_local_min;
    int nb_local_min_tot;
    int freeze_loc_min;
    int reset_limit;
    int nb_var_to_reset;
    int freeze_swap;
    int restart_max;
    int restart;
    int prob_select_loc_min;
    int first_best;
    int nb_reset;
    int nb_reset_tot;
    int reset_percent;
    
    int base_value;

    int data32[4];
    long long data64[2];          /* some 64 bits  */

} AdData;

__shared__ AdData p_ad;


__device__ void Select_Var_Min_Conflict_Parallel(void); 

__device__ void Select_Var_High_Cost(void);
__device__ void Select_Var_Min_Conflict(void); 


__device__ void Select_Var_High_Cost_Par(void);
__device__ void Select_Var_Min_Conflict_Par(void); 


__device__ void Select_Vars_To_Swap(void);
__device__ void Select_Vars_To_Swap_Par(void);

__device__ void Do_Reset(int);
__device__ int Reset(int);

__device__ int Cost(void);
__device__ int Cost_Seq(void);
__device__ int Cost_Of_Solution(int);
__device__ int Cost_On_Variable(int);
__device__ void Executed_Swap(int, int);
__device__ int Cost_If_Swap(int, int, int);

__device__ void Ad_Swap(int, int);

__device__ int Ad_Solve(int *);
__device__ void print_stat(void);

__shared__ int Buffer_Vals[THREADS];

#define isMaster  (threadIdx.x == 0)
//#define isMaster (tid == 0)


//#include "cap_tmp.cu"
//#include "magic-square.cu"
//#include "all-interval.cu"
#include "partit.cu"


__shared__ int do_reset;

__shared__ int use_local_min;
__shared__ int stop_var_sel;
__shared__ int list_len_j;
__shared__ int list_len_i;
__shared__ int buffer_tmp;

__device__ int Ad_Solve(int *winner) {

    int nb_in_plateau, best_of_best;
    
    if( isMaster ) {
      p_ad.nb_iter_tot = 0;
      p_ad.nb_swap_tot = 0;
      p_ad.nb_reset_tot = 0;
      p_ad.nb_restart = 0;
      
      p_ad.nb_iter = 0;
      p_ad.nb_swap = 0;
      p_ad.swap = 0;
      p_ad.nb_same_var = 0;
      p_ad.nb_local_min = 0;
      list_len_i = 0;
    }

restart:
    __syncthreads();
    if( isMaster ) {
      p_ad.nb_iter_tot+=p_ad.nb_iter;
      p_ad.nb_swap_tot+=p_ad.nb_swap;
      p_ad.nb_reset_tot+=p_ad.nb_reset;
      p_ad.nb_local_min_tot+=p_ad.nb_local_min;

      p_ad.nb_restart++;

      p_ad.nb_iter = 0;
      p_ad.nb_swap = 0;
      p_ad.swap = 0;
      p_ad.nb_same_var = 0;
      p_ad.nb_local_min = 0;
      list_len_i = 0;
    }
    //printf("x: %d\n",threadIdx.x);
    
#if __CUDA_ARCH__ >=200
    if( isMaster && blockIdx.x == 0 && p_ad.nb_restart==1) {
      printf("freeze local min: %d\nfreeze_swap: %d\n", p_ad.freeze_loc_min, p_ad.freeze_swap);
      printf("reset limit: %d\n", p_ad.reset_limit);
      printf("reset perc: %d\n", p_ad.reset_percent);
    }
#endif

    if( isMaster ) {
      int i;
      Random_Permut(p_ad.sol, p_ad.size, p_ad.base_value);
      //for(i=0;i<p_ad.size;i++) p_ad.sol[i]=i+1;

      //the number partition problem does not require a tabu list
      //for(i=0;i<p_ad.size;i++) p_ad.mark[i]=0;
      //memset(p_ad.mark, 0, p_ad.size * sizeof(unsigned));
      
      nb_in_plateau = 0;
      best_of_best = BIG;
      
      //p_ad.best_cost = p_ad.total_cost = Cost_Of_Solution(1);
    }
    
    __syncthreads();
    
    if( isMaster ) {
      p_ad.min_j = 0;
      int tmp_cost = Cost_Of_Solution(1);
      p_ad.best_cost = p_ad.total_cost = tmp_cost;
      //printf("Initial Cost[%d]: %d\n", blockIdx.x, p_ad.best_cost);
    }

    __syncthreads();
    while( (p_ad.total_cost!=0) && winner[0]==-1 && (p_ad.nb_iter < 100 && p_ad.nb_restart<1000000)) {
      //if( isMaster && p_ad.nb_iter % 100 == 0 )  {
      /*if( isMaster && blockIdx.x == 1) {
	printf("Iter[%d]: %d -- (%d, %d) -- swaps: %d, mark vars: %d -- list_len: %d, buffer_tmp[%d]: %d, local min: %d, nb_plateau: %d\n", p_ad.nb_iter, p_ad.total_cost, p_ad.max_i, p_ad.min_j, p_ad.nb_swap, p_ad.nb_var_marked, list_len_i, buffer_tmp, p_ad.list_i2_nb[buffer_tmp], p_ad.nb_local_min, nb_in_plateau);
	//printf("Marked max_i: %d -- min_j: %d\n", p_ad.mark[p_ad.max_i], p_ad.mark[p_ad.min_j]);
	}*/

      /*if( isMaster ) {
	printf("Iter[%d, %d]: %d (%d, %d), nb_var_marked: %d, nb_swap: %d, winner[0]: %d, winner[1]: %d\n", blockIdx.x, (p_ad.nb_iter+p_ad.nb_iter_tot), p_ad.total_cost, p_ad.max_i, p_ad.min_j, p_ad.nb_var_marked, p_ad.nb_swap, winner[0], winner[1]);
	}*/
      __syncthreads();
      
      if( isMaster ) {
	//printf("iter: %d \n",p_ad.nb_iter);
	if(p_ad.best_cost < best_of_best)
	  best_of_best = p_ad.best_cost;
	
	p_ad.nb_iter++;
      }
      
      //uncomment to use restarts or to stop the algorithm after a given number of iterations
      __syncthreads();
      if(p_ad.nb_iter >= p_ad.restart_limit) {
	if(p_ad.restart < p_ad.restart_max) 
	  goto restart;
	break;
      }
      
      __syncthreads();
      
#ifdef PAR
	Select_Vars_To_Swap_Par();
#else
	Select_Vars_To_Swap();
#endif
	__syncthreads();
	//if(isMaster) {printf("ITER: %d, var_marked: %d\n", p_ad.nb_iter, p_ad.nb_var_marked); }
	
	if(p_ad.min_j == -1)
	  continue;
        
	if( isMaster ) {
	  /*printf("total_cost: %d -- new_cost: %d\n",
	    p_ad.total_cost, p_ad.new_cost);*/
	    if(p_ad.total_cost != p_ad.new_cost) {
	      nb_in_plateau = 0;
	    }

	    if(p_ad.new_cost < p_ad.best_cost)
	      p_ad.best_cost = p_ad.new_cost;

	    nb_in_plateau++;
	}

	__syncthreads();

	//if(threadIdx.x == 0) {
	if(p_ad.max_i == p_ad.min_j) {
	  __syncthreads();
	  if( isMaster ) {
	    do_reset=0;
	    p_ad.nb_local_min++;
	    //Mark variable
	    Mark(p_ad.max_i, p_ad.freeze_loc_min);
	    if(p_ad.nb_var_marked + 1 >= p_ad.reset_limit) {
	      do_reset=1;
	      //Do_Reset(p_ad.nb_var_to_reset);
	    }
	  }
	  __syncthreads();
	  if(do_reset) {
	    Do_Reset(p_ad.nb_var_to_reset);
	  }
	  // __syncthreads();
	}
	else {
	    //__syncthreads();
	    if( isMaster ) {
		Mark(p_ad.max_i, p_ad.freeze_swap);
		Mark(p_ad.min_j, p_ad.freeze_swap);
		Ad_Swap(p_ad.max_i, p_ad.min_j);
		//printf("Marking variables: %d, %d\n",p_ad.mark[p_ad.max_i], p_ad.mark[p_ad.min_j]);
	    }
	    __syncthreads();

	    Executed_Swap(p_ad.max_i, p_ad.min_j);
	    
	    __syncthreads();

	    if( isMaster ) {
		p_ad.total_cost = p_ad.new_cost;
	    }

	    //__syncthreads();
	}
	//}
	__syncthreads();
    }

#if __CUDA_ARCH__ >= 200
    if( threadIdx.x == 0 && 0) {
      //printf("final cost: %d \n",p_ad.total_cost);
      print_stat();
      printf("WIN: %d\n", winner[0]);
    }
#endif

    if( winner[0] < 0 && isMaster /*&& p_ad.total_cost == 0*/) {
	int tmp = atomicExch(&winner[0], blockIdx.x);
	//in case that many threads finish nearly at the same time, only the winner will have access to this var
	if(tmp == -1) {
	  //Check_Solution(p_ad.sol, p_ad.size);
	    winner[1] = blockIdx.x;
	    winner[2] = p_ad.total_cost;
	    winner[3] = p_ad.nb_iter + p_ad.nb_iter_tot;
	    winner[4] = p_ad.nb_swap + p_ad.nb_swap_tot;
	    winner[5] = p_ad.nb_reset + p_ad.nb_reset_tot;
	    winner[6] = p_ad.nb_restart;
	    //printf("WINNER: %d, block.id: %d (%d), cost: %d, iter: %d, swaps: %d, resets: %d\n", winner[1], blockIdx.x, tmp,  p_ad.total_cost, p_ad.nb_iter, p_ad.nb_swap, p_ad.nb_reset);
	    printf("winner--[0]: %d\n", winner[0]);
	    //print_stat();
	    //__threadfence();
	    //asm("trap;");
	}
    }

    __syncthreads();

    return p_ad.total_cost;
}


#ifdef PAR
__shared__ int Buffer_Cost[THREADS];
__shared__ int Buffer_Vars[THREADS];


__device__ void 
Select_Vars_To_Swap_Par(void) {
  int s2 = size2*size2;
  int tid = threadIdx.x;
  int thread_size = Div_Round_Up(s2, THREADS);
  int init = tid * thread_size;
  int end  = init + thread_size;

  Buffer_Cost[ tid ] = p_ad.total_cost;
  Buffer_Vars[ tid ] = -1;

  p_ad.list_ij2_nb[tid ] = 0;

  int tmp;
  int i, j;
  
  //printf("TID[%d] init: %d, end: %d\n", tid, init, end);
  __syncthreads();

  for(int index=init; (index<end && index<s2); index++) {
    //1d to 2d mapping
    i = index / size2;
    j = index % size2 + size2;
    
    tmp = Cost_If_Swap(p_ad.total_cost, i, j);
    //printf("thread[%d], index: %d, i: %d, j: %d --> %d\n", tid, index, i, j);

    if( Buffer_Cost[tid] >= tmp ) {
      if( Buffer_Cost[tid] > tmp ) {
	Buffer_Cost[tid] = tmp;
	p_ad.list_ij2_nb[ tid ] = 0;
      }
      /*
	p_ad.list_ij2[ tid ][ p_ad.list_ij2_nb[tid] ].i = i;
	p_ad.list_ij2[ tid ][ p_ad.list_ij2_nb[tid] ].j = j;
	p_ad.list_ij2_nb[tid]++;
      */

      /*p_ad.list_ij2[ tid ][ p_ad.list_ij2_nb[tid]++ ] = index;
	if(p_ad.list_ij2_nb[tid] >= p_ad.size/THREADS) 
	p_ad.list_ij2_nb[tid] = 0;*/
      
      p_ad.list_ij2[ tid ][ p_ad.list_ij2_nb[tid] ] = index;
      p_ad.list_ij2_nb[tid] = (p_ad.list_ij2_nb[tid] + 1) % (p_ad.size / THREADS);
    }
  }

  if( p_ad.list_ij2[ tid ] == 0 )
    Buffer_Vars[ tid ] = -2;

  __syncthreads();
  //printf("tid[%d] init: %d, end: %d\n", tid, init, end);
  //__syncthreads();
  //asm("trap;");
  if( isMaster ) {
    int list_len = 0;
    int compute_vars = 1;
    p_ad.new_cost = p_ad.total_cost;
    for(int i=0;i<THREADS;i++) {
      if( p_ad.new_cost  >= Buffer_Cost[i]  && Buffer_Vars[i] != -2 ) {
	if( p_ad.new_cost > Buffer_Cost[i] ) {
	  p_ad.new_cost = Buffer_Cost[i];
	  list_len = 0;
	}
	p_ad.list_ij2[ THREADS ][ list_len++ ] = i;
      }
    }
    
    if( p_ad.new_cost >= p_ad.total_cost  ) {
      if( list_len == 0 || ( USE_PROB_SELECT_LOC_MIN && Random(100) <(unsigned) p_ad.prob_select_loc_min) ) {
	// 1 -> p_ad.nb_var_marked 
	p_ad.max_i = p_ad.min_j = 1;
	compute_vars = 0;
      }
      //else if( !USE_PROB_SELECT_LOC_MIN && (tmp = Random
    }

    if( compute_vars ) {
      int tmp2 = Random(list_len);
      int buffer_tmp = p_ad.list_ij2[THREADS][tmp2];
      
      tmp = Random( p_ad.list_ij2_nb[buffer_tmp] );
      /*p_ad.max_i = p_ad_list_ij2[ tmp ].i;
	p_ad.min_j = p_ad.list_ij2[ tmp ].j;*/
      int index2= p_ad.list_ij2[buffer_tmp][ tmp ];
      p_ad.max_i = index2 / size2;
      p_ad.min_j = index2 % size2 + size2;
    }
  }
}

#else

__device__ void
Select_Vars_To_Swap(void) {
  int i, j;
  int x;
  

  if( isMaster ) {
    
    p_ad.list_ij_nb = 0;
    p_ad.new_cost = BIG;
    p_ad.nb_var_marked = 0;
    stop_var_sel=0;
    
    for(i=0; i<size2; i++) {
      if(Marked(i)) {
	p_ad.nb_var_marked++;
	continue;
      }
      for(j=size2; j< p_ad.size; j++) {
      //for(j=i+1; j<p_ad.size; j++) {
	if(Marked(j)) continue;
	x = Cost_If_Swap(p_ad.total_cost, i, j);
	//printf("XX[%d,%d]: %d\n", i,j,x);
	if( x<= p_ad.new_cost ) {
	  if( x < p_ad.new_cost ) {
	    p_ad.new_cost = x;
	    p_ad.list_ij_nb = 0;
	    p_ad.list_ij[ p_ad.list_ij_nb ].i = i;
	    p_ad.list_ij[ p_ad.list_ij_nb ].j = j;
	    
	    p_ad.list_ij_nb = (p_ad.list_ij_nb + 1) % p_ad.size;
	    /*if( x < p_ad.total_cost) {
	      stop_var_sel=1;
	      goto a;
	      }*/
	  }
	}
      }
    }
  a:
    __syncthreads();
    if(stop_var_sel) {
      p_ad.max_i = p_ad.list_ij[0].i;
      p_ad.min_j = p_ad.list_ij[0].j;
      return;
    }
    int compute_vars = 1;
    
    if( p_ad.new_cost >= p_ad.total_cost ) {
      if( p_ad.list_ij_nb == 0 ||
	  (USE_PROB_SELECT_LOC_MIN && Random(100) < (unsigned) p_ad.prob_select_loc_min) ) {
	p_ad.max_i = p_ad.min_j = i;
	compute_vars = 0;
      }
      else if( !USE_PROB_SELECT_LOC_MIN && (x = Random(p_ad.list_ij_nb + p_ad.size)) < p_ad.size ) {
	p_ad.max_i = p_ad.min_j = x;
	compute_vars = 0;
      }
    }

    if( compute_vars ) {
      x = Random(p_ad.list_ij_nb);
      p_ad.max_i = p_ad.list_ij[x].i;
      p_ad.min_j = p_ad.list_ij[x].j;
    }

  }
  
}

#endif

__device__ void Ad_Swap(int i, int j) {
    int x;
    p_ad.nb_swap++;
    x = p_ad.sol[i];
    p_ad.sol[i] = p_ad.sol[j];
    p_ad.sol[j] = x;
}

//Only works for the deafult reset function.... other reset functions might require some modificaitons
//Be aware that in this case I am not unmarking varibles to avoid accessing global varaibles... One could just increase nb_swap+=[size of tabu list] to unmark those variables
__device__ void Do_Reset(int n) {
    //int cost = Reset(n);
    int cost;

    if( isMaster ) {
      // printf("Do reset\n");
	cost=Reset(n);
	p_ad.nb_reset++;
    }

    __syncthreads();

    if( isMaster ) {
      cost = Cost_Of_Solution(1);
      //printf("Reset...\n");
      p_ad.total_cost = cost;
      //for(int i=0;i<p_ad.size;i++) UnMark(i);
    }
    __syncthreads();
    //p_ad.total_cost = (cost < 0) ? Cost_Of_Solution(1) : cost;
}

__shared__ int iReset, jReset, xReset;
__shared__ int sizeReset;

__device__ int Reset1(int n) {
    int i, j, x;
    int size = p_ad.size;
 
    //sizeReset = p_ad.size;

    while(n--) {
        i = Random(size);
        j = Random(size);
        
        p_ad.nb_swap++;
        
        x = p_ad.sol[i];
        p_ad.sol[i] = p_ad.sol[j];
        p_ad.sol[j] = x;
    }

    return -1;
}


__shared__ hiprandState global_cudaRand;


__device__ void init_param(void) {

    if( isMaster ) {
    p_ad.size = SIZE;
    p_ad.nb_var_to_reset = -1;

    
    p_ad.restart_max = 0;


    //p_ad.seed = 12311;

    p_ad.prob_select_loc_min = 50;
    p_ad.freeze_loc_min = 1;
    p_ad.freeze_swap = 0;
    p_ad.reset_limit = 1;
    //p_ad.reset_percent = 5;
    p_ad.restart_limit = 1000000000;
    p_ad.restart_max = 0;
    p_ad.first_best = 0;

    
    if( p_ad.size < 8 || p_ad.size % 4 != 0 ) {
      printf("No solution with size = %d\n'",p_ad.size);
      asm("trap;");
    }

    sum_mid_x = (p_ad.size * (p_ad.size + 1)) / 4;
    sum_mid_x2 = ((long long) sum_mid_x * (2 * p_ad.size + 1)) / 3LL;
    coeff = sum_mid_x2 / sum_mid_x;
    
#if __CUDA_ARCH__ >= 200
    if(blockIdx.x == 0) {
      printf("mid sum x = %d,  mid sum x^2 = %lld, coeff: %d\n",
	     sum_mid_x, sum_mid_x2, coeff);
    }
#endif

    p_ad.prob_select_loc_min = 80;
    p_ad.freeze_loc_min = 1;
    p_ad.freeze_swap = 0;
    //    p_ad.reset_limit = SIZE_S2 * 1.2;
    p_ad.reset_percent = 1;
    p_ad.restart_limit = 100;
    p_ad.restart_max = 1000000;

    /*p_ad.prob_select_loc_min = 80;
    p_ad.freeze_loc_min = 4;
    p_ad.freeze_swap = 0;
    p_ad.reset_limit = 2;
    p_ad.reset_percent = 5;*/

    p_ad.base_value = 1;

    p_ad.nb_var_to_reset = Div_Round_Up(p_ad.size * p_ad.reset_percent, 100);

    if (p_ad.nb_var_to_reset < 2) {
	  p_ad.nb_var_to_reset = 2;
	  //printf("increasing nb var to reset since too small, now = %d\n", p_ad.nb_var_to_reset);
    }

    hiprand_init(p_ad.seed,0, 0, &global_cudaRand);
    int i;
    for(i=0;i<=blockIdx.x;i++) {
      p_ad.seed=(int)(hiprand_uniform(&global_cudaRand) * 100000.0);
    }
    //printf("Block[%d] -- seed: %d\n",blockIdx.x,p_ad.seed);
    hiprand_init(p_ad.seed, 0, 0, &cudaRand);

    }
}

__device__ void print_stat(void) {
#if __CUDA_ARCH__ >= 200
  if( isMaster ) {
    printf("block[%d] ->NB_iter: %d, local min: %d, swaps: %d, resets: %d, restarts: %d, cost: %d\n", blockIdx.x,
	   p_ad.nb_iter+p_ad.nb_iter_tot, p_ad.nb_local_min, p_ad.nb_swap+p_ad.nb_swap_tot, p_ad.nb_reset+p_ad.nb_reset_tot, p_ad.restart, p_ad.total_cost);
    //printf("Winner: %d\n",blockIdx.x);
  }
#endif
}

__device__ void sol2device( int *sol ) {
    
  if( isMaster ) {
    //printf("sol2device\n");
    for(int i=0;i<p_ad.size;i++) {
      sol[i]=p_ad.sol[i];
      //printf("SolD[%d]: %d -- %d\n",i,p_ad.sol[i], sol[i]);
    }
  }
  __syncthreads();
}

__global__ void main2(int seed, int *sol_device, int *winner) {

    p_ad.seed = seed;
    init_param();
    Solve(winner);

    if( isMaster && p_ad.total_cost == 0 && winner[1] == blockIdx.x) {
	print_stat();
	sol2device(sol_device);
	//printf("Winner: %d\n", winner[1] );
    }

    //printf("ENDING %d\n",threadIdx.x);

    __syncthreads();

}


#ifndef NBLOCK
#define NBLOCK 1
#endif

void sol2host(int *sol_device) {
    //int sol_host[SIZE];
    int *sol_host = (int*) malloc( SIZE * sizeof(int) );
    hipMemcpy(sol_host, sol_device, SIZE * sizeof(int), hipMemcpyDeviceToHost); 
    /*for(int i=0;i<SIZE;i++) {
	printf("Sol[%d]: %d\n",i, sol_host[i]);
    }*/
    Check_Solution(sol_host, SIZE);
    //Display_Solution(sol_host, SIZE);
}


inline void check_cuda_errors(const char *filename, const int line_number)
{

  hipDeviceSynchronize();
  hipError_t error = hipGetLastError();
  if(error != hipSuccess)
  {
    printf("CUDA error at %s:%i: %s\n", filename, line_number, hipGetErrorString(error));
    exit(-1);
  }

}


int main() {
#ifdef PAR
  printf("Parallel Neighbor Exploration On\n");
#else
  printf("Parallel Neighbor Exploration Off\n");
#endif
  time_t start, end;
  double length;
  time(&start);

  int *sol, *winner;
  
  //int w[2]={-1,-1};
  //w[0..1] reserved for winner information
  //Winner info
  //w[2] cost, w[3] nb_iters, w[4]  nb_swaps, w[5] nb_resets, w[6] nb_restarts
  int w[7] = {-1, -1, -1, -1, -1, -1, -1};
  
  size_t limit = 0;
  /*cudaDeviceGetLimi(&limit, hipLimitStackSize);
    printf("hipLimitStackSize: %u\n", (unsigned)limit); 
  */
  //    cudaThreadSetLimit(hipLimitMallocHeapSize, 512);
  //hipDeviceSetLimit(hipLimitMallocHeapSize, 128*1024*1024);
  hipDeviceSetLimit(hipLimitMallocHeapSize, 1024*1024*1024);
  hipDeviceSetLimit(hipLimitPrintfFifoSize, 1024*1024*100);
  hipDeviceGetLimit(&limit, hipLimitMallocHeapSize);
  printf("cudaLimitHeapSize: %u\n", (unsigned)limit);
  hipMalloc( (void**)&sol, SIZE * sizeof(int) );
  hipMalloc( (void**)&winner, 7*sizeof(int) );
  
  
  hipMemcpy( winner, w, 7*sizeof(int), hipMemcpyHostToDevice );
  

  int *sol_host = (int*) malloc( SIZE * sizeof(int) );
  for(int i=0;i<SIZE;i++) sol_host[i]=-1;
  
  hipMemcpy( sol, sol_host, SIZE * sizeof(int), hipMemcpyHostToDevice);
  int seed=time(NULL);
  //int seed = 1372314800;
  //int seed=1367568510;
  //int seed=368594130;
  //int seed=1372404090;
  printf("Seed: %d -- Threads: %d\n",seed, THREADS);
  printf("Dimensions, blocks: %d, threads: %d\n", NBLOCK, THREADS);
  main2<<<NBLOCK, THREADS>>>(seed, sol, winner);
  
  hipDeviceSynchronize();
  check_cuda_errors(__FILE__, __LINE__);
  //checkCUDAError("TEST");
  hipMemcpy(sol_host, sol, SIZE * sizeof(int), hipMemcpyDeviceToHost); 
  hipMemcpy(w, winner, 7*sizeof(int), hipMemcpyDeviceToHost);
  
  hipFree(sol);
  hipFree(winner);
  
  time(&end);
  length = difftime(end, start);
  printf("time: %.0f secs -- WINNER[%d], cost: %d, iter: %d, swaps: %d, resets: %d, restarts: %d\n", length, w[1], w[2], w[3], w[4], w[5], w[6]);

  //  printf("Time: %.0f secs\n", length);
  //hipDeviceReset();
  
  if( w[2] == 0 ) {
    Check_Solution(sol_host, SIZE);
    /*for(int i=0; i < SIZE/2; i++) 
      printf("%d ", sol_host[i]);
    printf("\n");
    for(int i=SIZE/2;i<SIZE;i++) 
      printf("%d ", sol_host[i]);
      printf("\n");*/
    //Display_Solution(sol_host, SIZE);
  }
  else {
    printf("NO SOLUTION FOUND!\n");
  }
  
  printf("Ending execution\n");
  return 1;
}
