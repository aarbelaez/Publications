/*
 *  Adaptive search - GPU Version
 *
 *
 * Please visit the https://pauillac.inria.fr/~diaz/adaptive/manual/index.html for a complete version of the original Adaptive Search code
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char** argv) {

      size_t limit = 0;

      hipDeviceGetLimit(&limit, hipLimitStackSize);
      printf("cudaLimitStackSize: %u\n", (unsigned)limit);
      hipDeviceGetLimit(&limit, hipLimitPrintfFifoSize);
      printf("cudaLimitPrintfFifoSize: %u\n", (unsigned)limit);
      hipDeviceGetLimit(&limit, hipLimitMallocHeapSize);
      printf("cudaLimitMallocHeapSize: %u\n", (unsigned)limit);

      limit = 9999;
      
      hipDeviceSetLimit(hipLimitStackSize, limit);
      hipDeviceSetLimit(hipLimitPrintfFifoSize, limit);
      hipDeviceSetLimit(hipLimitMallocHeapSize, limit);

      limit = 0;

      hipDeviceGetLimit(&limit, hipLimitStackSize);
      printf("New cudaLimitStackSize: %u\n", (unsigned)limit);
      hipDeviceGetLimit(&limit, hipLimitPrintfFifoSize);
      printf("New cudaLimitPrintfFifoSize: %u\n", (unsigned)limit);
      hipDeviceGetLimit(&limit, hipLimitMallocHeapSize);
      printf("New cudaLimitMallocHeapSize: %u\n", (unsigned)limit);

      return 0;
}
