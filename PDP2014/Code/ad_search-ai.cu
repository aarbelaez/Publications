#include "hip/hip_runtime.h"
/*
 *  Adaptive search - GPU Version
 *
 *
 * Please visit the https://pauillac.inria.fr/~diaz/adaptive/manual/index.html for a complete version of the original Adaptive Search code
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include<hiprand/hiprand_kernel.h>
#include <time.h>

#include "mrand.cu"


#define BIG ((unsigned int) -1 >> 1) 

#define BASE_MARK    ((unsigned) p_ad.nb_swap)
#define Mark(i, k)   p_ad.mark[i] = BASE_MARK + (k)
#define UnMark(i)    p_ad.mark[i] = 0
#define Marked(i)    (BASE_MARK + 1 <= p_ad.mark[i])

#define USE_PROB_SELECT_LOC_MIN ((unsigned) p_ad.prob_select_loc_min <= 100)

#define Div_Round_Up(x, y)   (((x) + (y) - 1) / (y))

#ifndef SIZE
#define SIZE 10
#endif

#ifndef THREADS
#define THREADS 2
#endif

//#define Arrayaccess(a, i, j) ((a)[(i) * SIZE + (j)])
#define Arrayaccess(i,j) ( (i) * SIZE + (j) )

typedef struct
{
  int i, j;
}Pair;

typedef struct {
    int max_i;
    int min_j;
    int new_cost;
    int best_cost;
  //int *mark;
  int mark[SIZE];
    int nb_var_marked;
    int size;
    int seed;

    //int list_i[SIZE];
    int *list_i;
    int list_i_nb;

    //int list_j[SIZE];
    int *list_j;
    int list_j_nb;

  //int list_i2_nb[THREADS];
  //int list_j2_nb[THREADS];
    /*int *list_j2[THREADS];
    int *list_i2[THREADS];*/
    //int **list_j2;
    //int **list_i2;
    int list_ij_nb;
    Pair list_ij[ SIZE ];
    
    int sol[SIZE];
    //int *sol;
    int total_cost;
    
    int nb_iter;
    int restart_limit;
    int nb_restart;
    int nb_swap;
    int swap;
    int nb_same_var;
    int nb_local_min;
    int freeze_loc_min;
    int reset_limit;
    int nb_var_to_reset;
    int freeze_swap;
    int restart_max;
    int restart;
    int prob_select_loc_min;
    int first_best;
    int nb_reset;
    int reset_percent;
    
    int base_value;

    int data32[4];

} AdData;

__shared__ AdData p_ad;


__device__ void Select_Var_Min_Conflict_Parallel(void); 

__device__ void Select_Var_High_Cost(void);
__device__ void Select_Var_Min_Conflict(void); 


__device__ void Select_Var_High_Cost_Par(void);
__device__ void Select_Var_Min_Conflict_Par(void); 


__device__ void Select_Vars_To_Swap(void);
__device__ void Select_Vars_To_Swap_Par(void);

__device__ void Do_Reset(int);
__device__ int Reset(int);

__device__ int Cost(void);
__device__ int Cost_Seq(void);
__device__ int Cost_Of_Solution(int);
__device__ int Cost_On_Variable(int);
__device__ void Executed_Swap(int, int);
__device__ int Cost_If_Swap(int, int, int);

__device__ void Ad_Swap(int, int);

__device__ int Ad_Solve(int *);

__shared__ int Buffer_Vals[THREADS];

#define isMaster  (threadIdx.x == 0)
//#define isMaster (tid == 0)


//#include "cap_tmp.cu"
//#include "magic-square.cu"
#include "all-interval.cu"


__shared__ int do_reset;

__shared__ int use_local_min;
__shared__ int stop_var_sel;
__shared__ int list_len_j;
__shared__ int list_len_i;
__shared__ int buffer_tmp;

__device__ int Ad_Solve(int *winner) {

    int nb_in_plateau, best_of_best;

    if( isMaster ) {
	p_ad.nb_restart=-1;
	p_ad.nb_iter = 0;
	p_ad.nb_swap = 0;
	p_ad.swap = 0;
	p_ad.nb_same_var = 0;
	p_ad.nb_restart = 0;
	p_ad.nb_local_min = 0;
	list_len_i = 0;
    }
    //printf("x: %d\n",threadIdx.x);
restart:

#if __CUDA_ARCH__ >=200
    if( isMaster && blockIdx.x == 0) {
      printf("freeze local min: %d\nfreeze_swap: %d\n", p_ad.freeze_loc_min, p_ad.freeze_swap);
      printf("reset limit: %d\n", p_ad.reset_limit);
      printf("reset perc: %d\n", p_ad.reset_percent);
    }
#endif

    if( isMaster ) {
      int i;
      Random_Permut(p_ad.sol, p_ad.size, p_ad.base_value);

      for(i=0;i<p_ad.size;i++) p_ad.mark[i]=0;
      //memset(p_ad.mark, 0, p_ad.size * sizeof(unsigned));
      
      nb_in_plateau = 0;
      p_ad.nb_restart++;
      best_of_best = BIG;
      
      //p_ad.best_cost = p_ad.total_cost = Cost_Of_Solution(1);
    }
    
    __syncthreads();
    
    if( isMaster ) {
      p_ad.min_j = 0;
      int tmp_cost = Cost_Of_Solution(1);
      p_ad.best_cost = p_ad.total_cost = tmp_cost;
      //printf("Initial Cost[%d]: %d\n", blockIdx.x, p_ad.best_cost);
    }

    __syncthreads();
    while( (p_ad.total_cost!=0) && winner[0]==-1 && p_ad.nb_iter < 200 ) {
      //if( isMaster && p_ad.nb_iter % 100 == 0 )  {
      /*if( isMaster && blockIdx.x == 1) {
	printf("Iter[%d]: %d -- (%d, %d) -- swaps: %d, mark vars: %d -- list_len: %d, buffer_tmp[%d]: %d, local min: %d, nb_plateau: %d\n", p_ad.nb_iter, p_ad.total_cost, p_ad.max_i, p_ad.min_j, p_ad.nb_swap, p_ad.nb_var_marked, list_len_i, buffer_tmp, p_ad.list_i2_nb[buffer_tmp], p_ad.nb_local_min, nb_in_plateau);
	//printf("Marked max_i: %d -- min_j: %d\n", p_ad.mark[p_ad.max_i], p_ad.mark[p_ad.min_j]);
	}*/

      /*if( isMaster ) {
	printf("Iter[%d]: %d (%d, %d), nb_var_marked: %d\n", p_ad.nb_iter, p_ad.total_cost, p_ad.max_i, p_ad.min_j, p_ad.nb_var_marked);
	}*/
      __syncthreads();

	if( isMaster ) {
	    //printf("iter: %d \n",p_ad.nb_iter);
	    if(p_ad.best_cost < best_of_best)
		best_of_best = p_ad.best_cost;

	    p_ad.nb_iter++;
	}

	//uncomment to use restarts or to stop the algorithm after a given number of iterations
	/*
        if(p_ad.nb_iter >= p_ad.restart_limit) {
            if(p_ad.restart < p_ad.restart_max) 
                goto restart;
            break;
        }
        */
	__syncthreads();

#ifdef PAR
	Select_Vars_To_Swap_Par();
#else
	Select_Vars_To_Swap();
#endif
	__syncthreads();
	//if(isMaster) {printf("ITER: %d, var_marked: %d\n", p_ad.nb_iter, p_ad.nb_var_marked); }
	
	if(p_ad.min_j == -1)
	  continue;
        
	if( isMaster ) {
	  /*printf("total_cost: %d -- new_cost: %d\n",
	    p_ad.total_cost, p_ad.new_cost);*/
	    if(p_ad.total_cost != p_ad.new_cost) {
	      nb_in_plateau = 0;
	    }

	    if(p_ad.new_cost < p_ad.best_cost) 
	      p_ad.best_cost = p_ad.new_cost;

	    nb_in_plateau++;
	}

	__syncthreads();

	//if(threadIdx.x == 0) {
	if(p_ad.max_i == p_ad.min_j) {
	  __syncthreads();
	  if( isMaster ) {
	    do_reset=0;
	    p_ad.nb_local_min++;
	    //Mark variable
	    Mark(p_ad.max_i, p_ad.freeze_loc_min);
	    if(p_ad.nb_var_marked + 1 >= p_ad.reset_limit) {
	      do_reset=1;
	      //Do_Reset(p_ad.nb_var_to_reset);
	    }
	  }
	  __syncthreads();
	  if(do_reset) {
	    Do_Reset(p_ad.nb_var_to_reset);
	  }
	    //__syncthreads();
	}
	else {
	    //__syncthreads();
	    if( isMaster ) {
		Mark(p_ad.max_i, p_ad.freeze_swap);
		Mark(p_ad.min_j, p_ad.freeze_swap);
		Ad_Swap(p_ad.max_i, p_ad.min_j);
		//printf("Marking variables: %d, %d\n",p_ad.mark[p_ad.max_i], p_ad.mark[p_ad.min_j]);
	    }
	    __syncthreads();

	    Executed_Swap(p_ad.max_i, p_ad.min_j);
	    
	    __syncthreads();

	    if( isMaster ) {
		p_ad.total_cost = p_ad.new_cost;
	    }

	    //__syncthreads();
	}
	//}
	__syncthreads();
    }

#if __CUDA_ARCH__ >= 200
    if( threadIdx.x == 0) {
      printf("final cost: %d \n",p_ad.total_cost);
    }
#endif

    if( winner[0] < 0 && isMaster /*&& p_ad.total_cost == 0*/) {
	int tmp = atomicExch(&winner[0], blockIdx.x);
	//in case that many threads finish nearly at the same time, only the winner will have access to this var
	if(tmp == -1) {
	    winner[1] = blockIdx.x;
	    winner[2] = p_ad.total_cost;
	    winner[3] = p_ad.nb_iter;
	    winner[4] = p_ad.nb_swap;
	    winner[5] = p_ad.nb_reset;
	    //printf("WINNER: %d, block.id: %d (%d), cost: %d, iter: %d, swaps: %d, resets: %d\n", winner[1], blockIdx.x, tmp,  p_ad.total_cost, p_ad.nb_iter, p_ad.nb_swap, p_ad.nb_reset);
	}
    }

    __syncthreads();

    return p_ad.total_cost;
}



/*__shared__ int Buffer_Cost[THREADS];
__shared__ int Buffer_Vars[THREADS];
*/


__device__ void
Select_Vars_To_Swap_Par(void) {
  int i, j;
  int x;
  
  if( isMaster ) {
    p_ad.list_ij_nb = 0;
    p_ad.new_cost = BIG;
    p_ad.nb_var_marked = 0;
    stop_var_sel=0;
  }

  
  for(i=0; i<p_ad.size; i++) {
    __syncthreads();
    if( Marked(i) ) { 
      if( isMaster ) {
	//printf("Marked: %d\n", i);
	p_ad.nb_var_marked++;
      }
      continue;
    }
    for(j=i+1; j<p_ad.size; j++) {
      __syncthreads();
      if( Marked(j) ) continue;
      x = Cost_If_Swap(p_ad.total_cost, i, j);
      __syncthreads();
      
      if( isMaster ) {
	if( x<= p_ad.new_cost ) {
	  if( x < p_ad.new_cost ) {
	    p_ad.new_cost = x;
	    p_ad.list_ij_nb = 0;
	    p_ad.list_ij[ p_ad.list_ij_nb ].i = i;
	    p_ad.list_ij[ p_ad.list_ij_nb ].j = j;
	    
	    p_ad.list_ij_nb = (p_ad.list_ij_nb + 1) % p_ad.size;
	    if( x < p_ad.total_cost )
	      stop_var_sel=1;
	  }
	}
      }
      __syncthreads();
      if(stop_var_sel) {
	//printf("HPAAA: cost: %d new_cost: %d\n", p_ad.total_cost, p_ad.new_cost);
	p_ad.max_i = p_ad.list_ij[0].i;
	p_ad.min_j = p_ad.list_ij[0].j;
	return;
      }
    }
  }
  
  __syncthreads();
  if( isMaster ) {
    int compute_vars = 1;
    
    if( p_ad.new_cost >= p_ad.total_cost ) {
      if( p_ad.list_ij_nb == 0 ||
	  (USE_PROB_SELECT_LOC_MIN && Random(100) < (unsigned) p_ad.prob_select_loc_min) ) {
	p_ad.max_i = p_ad.min_j = p_ad.nb_var_marked;
	compute_vars = 0;
      }
      else if( !USE_PROB_SELECT_LOC_MIN && (x = Random(p_ad.list_ij_nb + p_ad.size)) < p_ad.size ) {
	p_ad.max_i = p_ad.min_j = x;
	compute_vars = 0;
      }
    }
    
    if( compute_vars ) {
      x = Random(p_ad.list_ij_nb);
      p_ad.max_i = p_ad.list_ij[x].i;
      p_ad.min_j = p_ad.list_ij[x].j;
    }
  }
}


__device__ void
Select_Vars_To_Swap(void) {
  int i, j;
  int x;
  

  if( isMaster ) {
    
    p_ad.list_ij_nb = 0;
    p_ad.new_cost = BIG;
    p_ad.nb_var_marked = 0;
    stop_var_sel=0;
    
    for(i=0; i<p_ad.size; i++) {
      if(Marked(i)) {
	p_ad.nb_var_marked++;
	continue;
      }
      for(j=i+1; j<p_ad.size; j++) {
	if(Marked(j)) continue;
	x = Cost_If_Swap(p_ad.total_cost, i, j);
	//printf("XX[%d,%d]: %d\n", i,j,x);
	if( x<= p_ad.new_cost ) {
	  if( x < p_ad.new_cost ) {
	    p_ad.new_cost = x;
	    p_ad.list_ij_nb = 0;
	    p_ad.list_ij[ p_ad.list_ij_nb ].i = i;
	    p_ad.list_ij[ p_ad.list_ij_nb ].j = j;
	    
	    p_ad.list_ij_nb = (p_ad.list_ij_nb + 1) % p_ad.size;
	    if( x < p_ad.total_cost) {
	      stop_var_sel=1;
	      goto a;
	    }
	  }
	}
      }
    }
  a:
    __syncthreads();
    if(stop_var_sel) {
      p_ad.max_i = p_ad.list_ij[0].i;
      p_ad.min_j = p_ad.list_ij[0].j;
      return;
    }
    int compute_vars = 1;
    
    if( p_ad.new_cost >= p_ad.total_cost ) {
      if( p_ad.list_ij_nb == 0 ||
	  (USE_PROB_SELECT_LOC_MIN && Random(100) < (unsigned) p_ad.prob_select_loc_min) ) {
	p_ad.max_i = p_ad.min_j = i;
	compute_vars = 0;
      }
      else if( !USE_PROB_SELECT_LOC_MIN && (x = Random(p_ad.list_ij_nb + p_ad.size)) < p_ad.size ) {
	p_ad.max_i = p_ad.min_j = x;
	compute_vars = 0;
      }
    }
    
    if( compute_vars ) {
      x = Random(p_ad.list_ij_nb);
      p_ad.max_i = p_ad.list_ij[x].i;
      p_ad.min_j = p_ad.list_ij[x].j;
    }

  }
  
}

__device__ void Ad_Swap(int i, int j) {
    int x;
    p_ad.nb_swap++;
    x = p_ad.sol[i];
    p_ad.sol[i] = p_ad.sol[j];
    p_ad.sol[j] = x;
}

//Only works for the deafult reset function.... other reset functions might require some modificaitons
//Be aware that in this case I am not unmarking varibles to avoid accessing global varaibles... One could just increase nb_swap+=[size of tabu list] to unmark those variables
__device__ void Do_Reset(int n) {
    //int cost = Reset(n);
    int cost;

    if( isMaster ) {
      // printf("Do reset\n");
	cost=Reset(n);
	p_ad.nb_reset++;
    }

    __syncthreads();

    if( isMaster ) {
      cost = Cost_Of_Solution(1);
      //printf("Reset...\n");
      p_ad.total_cost = cost;
      //for(int i=0;i<p_ad.size;i++) UnMark(i);
    }
    __syncthreads();
    //p_ad.total_cost = (cost < 0) ? Cost_Of_Solution(1) : cost;
}

__shared__ int iReset, jReset, xReset;
__shared__ int sizeReset;

__device__ int Reset1(int n) {
    int i, j, x;
    int size = p_ad.size;
 
    //sizeReset = p_ad.size;

    while(n--) {
        i = Random(size);
        j = Random(size);
        
        p_ad.nb_swap++;
        
        x = p_ad.sol[i];
        p_ad.sol[i] = p_ad.sol[j];
        p_ad.sol[j] = x;
    }

    return -1;
}


__shared__ hiprandState global_cudaRand;


__device__ void init_param(void) {

    if( isMaster ) {
    p_ad.size = SIZE;
    p_ad.nb_var_to_reset = -1;

    
    p_ad.restart_max = 0;


    //p_ad.seed = 12311;

    p_ad.prob_select_loc_min = 50;
    p_ad.freeze_loc_min = 1;
    p_ad.freeze_swap = 0;
    p_ad.reset_limit = 1;
    //p_ad.reset_percent = 5;
    p_ad.restart_limit = 1000000000;
    p_ad.restart_max = 0;
    p_ad.first_best = 0;

    
    p_ad.prob_select_loc_min = 66;
    p_ad.freeze_loc_min = 1;
    p_ad.freeze_swap = 0;
    //    p_ad.reset_limit = SIZE_S2 * 1.2;
    p_ad.reset_percent = 10;
    p_ad.restart_limit = 10000000;
    p_ad.restart_max = 0;

    /*p_ad.prob_select_loc_min = 80;
    p_ad.freeze_loc_min = 4;
    p_ad.freeze_swap = 0;
    p_ad.reset_limit = 2;
    p_ad.reset_percent = 5;*/

    p_ad.base_value = 1;

    p_ad.reset_percent = 5;
    p_ad.nb_var_to_reset = Div_Round_Up(p_ad.size * p_ad.reset_percent, 100);

    if (p_ad.nb_var_to_reset < 2) {
	  p_ad.nb_var_to_reset = 2;
	  //printf("increasing nb var to reset since too small, now = %d\n", p_ad.nb_var_to_reset);
    }

    hiprand_init(p_ad.seed,0, 0, &global_cudaRand);
    int i;
    for(i=0;i<=blockIdx.x;i++) {
	p_ad.seed=(int)(hiprand_uniform(&global_cudaRand) * 100000.0);
    }
    //printf("Block[%d] -- seed: %d\n",blockIdx.x,p_ad.seed);
    hiprand_init(p_ad.seed, 0, 0, &cudaRand);

    }
}

__device__ void print_stat(void) {
#if __CUDA_ARCH__ >= 200
    if( isMaster ) {
	printf("NB_iter: %d, local min: %d, swaps: %d, resets: %d, cost: %d\n",
	      p_ad.nb_iter, p_ad.nb_local_min, p_ad.nb_swap, p_ad.nb_reset, p_ad.total_cost);
	//printf("Winner: %d\n",blockIdx.x);
    }
#endif
}

__device__ void sol2device( int *sol ) {
    
    if( isMaster ) {
      //printf("sol2device\n");
	for(int i=0;i<p_ad.size;i++) {
	    sol[i]=p_ad.sol[i];
	    //printf("SolD[%d]: %d -- %d\n",i,p_ad.sol[i], sol[i]);
	}
    }
    __syncthreads();
}

__global__ void main2(int seed, int *sol_device, int *winner) {

    p_ad.seed = seed;
    init_param();
    Solve(winner);

    if( isMaster && p_ad.total_cost == 0 && winner[1] == blockIdx.x) {
	print_stat();
	sol2device(sol_device);
	//printf("Winner: %d\n", winner[1] );
    }

    //printf("ENDING %d\n",threadIdx.x);

    __syncthreads();

}


#ifndef NBLOCK
#define NBLOCK 1
#endif

void sol2host(int *sol_device) {
    //int sol_host[SIZE];
    int *sol_host = (int*) malloc( SIZE * sizeof(int) );
    hipMemcpy(sol_host, sol_device, SIZE * sizeof(int), hipMemcpyDeviceToHost); 
    /*for(int i=0;i<SIZE;i++) {
	printf("Sol[%d]: %d\n",i, sol_host[i]);
    }*/
    Check_Solution(sol_host, SIZE);
    //Display_Solution(sol_host, SIZE);
}


inline void check_cuda_errors(const char *filename, const int line_number)
{

  hipDeviceSynchronize();
  hipError_t error = hipGetLastError();
  if(error != hipSuccess)
  {
    printf("CUDA error at %s:%i: %s\n", filename, line_number, hipGetErrorString(error));
    exit(-1);
  }

}


int main() {
#ifdef PAR
  printf("Parallel Neighbor Exploration On\n");
#else
  printf("Parallel Neighbor Exploration Off\n");
#endif
  time_t start, end;
  double length;
  time(&start);

  int *sol, *winner;
  
  //int w[2]={-1,-1};
  //w[0..1] reserved for winner information
  //Winner info
  //w[2] cost, w[3] nb_iters, w[4]  nb_swaps, w[5] nb_resets
  int w[6] = {-1, -1, -1, -1, -1, -1};
  
  size_t limit = 0;
  /*hipDeviceGetLimit(&limit, hipLimitStackSize);
    printf("hipLimitStackSize: %u\n", (unsigned)limit); 
  */
  //    cudaThreadSetLimit(hipLimitMallocHeapSize, 512);
  //hipDeviceSetLimit(hipLimitMallocHeapSize, 128*1024*1024);
  hipDeviceSetLimit(hipLimitMallocHeapSize, 1024*1024*1024);
  hipDeviceSetLimit(hipLimitPrintfFifoSize, 1024*1024*100);
  hipDeviceGetLimit(&limit, hipLimitMallocHeapSize);
  printf("cudaLimitHeapSize: %u\n", (unsigned)limit);
  hipMalloc( (void**)&sol, SIZE * sizeof(int) );
  hipMalloc( (void**)&winner, 6*sizeof(int) );
  
  
  hipMemcpy( winner, w, 6*sizeof(int), hipMemcpyHostToDevice );
  

  int *sol_host = (int*) malloc( SIZE * sizeof(int) );
  for(int i=0;i<SIZE;i++) sol_host[i]=-1;
  
  hipMemcpy( sol, sol_host, SIZE * sizeof(int), hipMemcpyHostToDevice);
  int seed=time(NULL);
  //int seed=1367568510;
  //int seed=368594130;
  //int seed=1371110766;
  printf("Seed: %d -- Threads: %d\n",seed, THREADS);
  printf("Dimensions, blocks: %d, threads: %d\n", NBLOCK, THREADS);
  main2<<<NBLOCK, THREADS>>>(seed, sol, winner);
  
  hipDeviceSynchronize();
  check_cuda_errors(__FILE__, __LINE__);
  //checkCUDAError("TEST");
  hipMemcpy(sol_host, sol, SIZE * sizeof(int), hipMemcpyDeviceToHost); 
  hipMemcpy(w, winner, 6*sizeof(int), hipMemcpyDeviceToHost);
  printf("WINNER[%d], cost: %d, iter: %d, swaps: %d, resets: %d\n", w[1], w[2], w[3], w[4], w[5]);
  
  hipFree(sol);
  hipFree(winner);
  
  time(&end);
  length = difftime(end, start);
  printf("Time: %.0f secs\n", length);
  //hipDeviceReset();
  
  if( w[2] == 0 ) {
    Check_Solution(sol_host, SIZE);
    for(int i=0; i < SIZE; i++) 
      printf("%d ", sol_host[i]);
    printf("\n");
    //Display_Solution(sol_host, SIZE);
  }
  else {
    printf("NO SOLUTION FOUND!\n");
  }
  
  printf("Ending execution\n");
  return 1;
}
